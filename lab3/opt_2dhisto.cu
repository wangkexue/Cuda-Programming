#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdlib.h>
#include <string.h>

#include <cutil.h>
#include "util.h"
#include "ref_2dhisto.h"

#define T 12

__global__ void opt_2dhistoKernel(uint32_t*, size_t, size_t, uint32_t*);
__global__ void opt_32to8Kernel(uint32_t*, uint8_t*, size_t);

void opt_2dhisto(uint32_t* input, size_t height, size_t width, uint8_t* bins, uint32_t* g_bins)
{
    /* This function should only contain a call to the GPU 
       histogramming kernel. Any memory allocations and
       transfers must be done outside this function */

    // Kernel to calculate the bins
    // We use 1024 * T threads so that more streaming multiprocessors can be used
    //opt_2dhistoKernel<<<2 * T, 512>>>(input, height, width, g_bins);
    /*
    dim3 block(16, 16);
    dim3 grid( ((INPUT_WIDTH + 128) & 0xFFFFFF80) / 16, INPUT_HEIGHT / 16);
    opt_2dhistoKernel<<<grid, block>>>(input, height, width, g_bins);    
    */
    //hipDeviceSynchronize();  
    opt_2dhistoKernel<<<INPUT_HEIGHT * ((INPUT_WIDTH + 128) & 0xFFFFFF80) / 1024 , 1024>>>(input, height, width, g_bins);

    // Convert 32 bit to 8 bit
    opt_32to8Kernel<<<HISTO_HEIGHT * HISTO_WIDTH / 512, 512>>>(g_bins, bins, 1024);

    hipDeviceSynchronize();
}

/* Include below the implementation of any other functions you need */
/* kernel verson 1: basic */
/*
__global__ void opt_2dhistoKernel(uint32_t *input, size_t height, size_t width, uint32_t* bins){
     int col = blockDim.x * blockIdx.x + threadIdx.x;
     int row = blockDim.y * blockIdx.y + threadIdx.y;
     if (row == 0 && col < 1024)
         bins[col] = 0;
     __syncthreads();
     if (row < height && col < width)
        atomicAdd(&bins[input[col + row * ((INPUT_WIDTH + 128) & 0xFFFFFF80)]], 1);
     //__syncthreads();
}
*/
/* kernel verson 2: stride */
__global__ void opt_2dhistoKernel(uint32_t *input, size_t height, size_t width, uint32_t* bins){
     int i = blockDim.x * blockIdx.x + threadIdx.x;
     if (i < 1024)
        bins[i] = 0;
     __syncthreads();
     int stride = blockDim.x * gridDim.x;
     while (i < 4096 * height)
     {
        if (i %  ((INPUT_WIDTH + 128) & 0xFFFFFF80) < width )
           atomicAdd( &(bins[input[i]]), 1 );
        i += stride;
     }
}


__global__ void opt_32to8Kernel(uint32_t *input, uint8_t* output, size_t length){
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	
	output[idx] = (uint8_t)((input[idx] < UINT8_MAX) * input[idx]) + (input[idx] >= UINT8_MAX) * UINT8_MAX;

	__syncthreads();
}

void* AllocateDevice(size_t size){
	void* ret;
	hipMalloc(&ret, size);
	return ret;
}

void CopyToDevice(void* D_device, void* D_host, size_t size){
	hipMemcpy(D_device, D_host, size, 
					hipMemcpyHostToDevice);
}

void CopyFromDevice(void* D_host, void* D_device, size_t size){
	hipMemcpy(D_host, D_device, size, 
					hipMemcpyDeviceToHost);
}

void FreeDevice(void* D_device){
	hipFree(D_device);
}
